
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int* a) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int answer = idx;
  int i = 0;

  if (answer != 0) {
    while (answer != 1) {
      if (answer % 2 == 0) { answer /= 2; }
      else { answer = 3 * answer + 1; }
      i++;
    }
  }

  a[idx] = i;
}

int main() {
  int dimx = 3907*256;
  int num_bytes = dimx * sizeof(int);

  int *d_a = 0, *h_a = 0;

  h_a = (int*)malloc(num_bytes);
  hipMalloc((void**)&d_a, num_bytes);

  if (0==h_a || 0==d_a) {
    printf("can't allocate memory");
  }

  hipMemset(d_a, 0, num_bytes);
  hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop);

  hipEventRecord(start, 0);
  kernel<<<3907, 256>>>(d_a);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  float et;
  hipEventElapsedTime(&et, start, stop);

  hipEventDestroy(start); hipEventDestroy(stop);

  printf("kernel execution time: %8.6fms\n", et);

  hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);

  int max = 0;
  for(int i=0; i<dimx; i++) {
//    printf("%d ", h_a[i]);
    if (h_a[i] > max) max = h_a[i];
  }
  printf("max is %d\n", max);

  free(h_a);
  hipFree(d_a); 

  return 0;
} 
