
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int* a) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
//  a[idx] = 7;
  a[idx] = blockIdx.x;
//  a[idx] = threadIdx.x;
}

int main() {

  int dc = -1;
  hipGetDeviceCount(&dc);
  printf("%d device(s) present\n", dc);

  int cd = -1;
  hipGetDevice(&cd);
  printf("device id used: %d\n", cd);

  int dimx = 32;
  int num_bytes = dimx * sizeof(int);

  int *d_a = 0, *h_a = 0;

  h_a = (int*)malloc(num_bytes);
  int rv = hipMalloc((void**)&d_a, num_bytes);

  if (0==h_a) { printf("couldn't allocate host memory\n"); }
  if (0==d_a) { printf("couldn't allocate device memory\n"); }

  hipMemset(d_a, 0, num_bytes);
  hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop);

  hipEventRecord(start, 0);
  kernel<<<8, 8>>>(d_a);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  float et;
  hipEventElapsedTime(&et, start, stop);

  hipEventDestroy(start); hipEventDestroy(stop);

  printf("kernel execution time: %8.6fms\n", et);

  hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);

  for(int i=0; i<dimx; i++) {
    printf("%d ", h_a[i]);
  }
  printf("\n");

  free(h_a);
  hipFree(d_a); 

  return 0;
} 
